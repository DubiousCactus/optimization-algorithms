#include <hip/hip_runtime.h>

#include <hipblas.h>
#include<stdio.h>


void printVector(const double* vector, int n)
{
    for(int i = 0; i < n; i++)
    {
        printf("%g", vector[i]);
        if(i != n-1)
        {
            printf(",");
        }
    }

    printf("\n");

}

void createHandle(hipblasHandle_t* handle)
{
	hipblasCreate(handle);
}

void destroyHandle(hipblasHandle_t* handle)
{
    hipblasDestroy(*handle);
}

void initCUDA()
{
	hipMalloc(NULL, 0);
}

int mallocCUDA(double** A_d, int nBytes)
{
    hipError_t error;

    error = hipMalloc(A_d, nBytes);

    if (error != hipSuccess)
    {
        return 1;
    }
    else
    {
        return 0;
    }
}

int memcpyCUDA(double* A_d, const double* A_h, int nBytes)
{
    hipError_t error;

    error = hipMemcpy(A_d, A_h, nBytes, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        return 1;
    }
    else
    {
        return 0;
    }
}

void freeCUDA(double* A_d)
{
	hipFree(A_d);
}

void dfpInverseHessian(
        double* old_parameters, double* parameters,
        double* old_gradient, double* gradient,
        double* old_inverse_Hessian, double* auxiliar_matrix,
        double* inverse_Hessian_host, int n)
{
	double alpha;
	double beta;
	
	double parameters_dot_gradient;
	double gradient_dot_Hessian_dot_gradient;
	
    double* vector = (double*)malloc(n*sizeof(double));

	hipblasHandle_t handle;
	
	createHandle(&handle);
	
	alpha = -1;

    hipblasDaxpy(handle, n, &alpha, old_parameters, 1, parameters, 1);
    hipblasDaxpy(handle, n, &alpha, old_gradient, 1, gradient, 1);

    hipblasDdot(handle, n, parameters, 1, gradient, 1, &parameters_dot_gradient);

    alpha = 1;
	beta = 0;
	
	hipblasDgemv(handle, HIPBLAS_OP_N, n, n, &alpha, old_inverse_Hessian, n, gradient, 1, &beta, old_gradient, 1);
	
    hipblasDdot(handle, n, gradient, 1, old_gradient, 1, &gradient_dot_Hessian_dot_gradient);
	
	alpha = 0;

    hipblasDscal(handle, n*n, &alpha, auxiliar_matrix, 1);
	
	alpha = 1;
	
	hipblasDger(handle, n, n, &alpha, parameters, 1, parameters, 1, auxiliar_matrix, n);
	
	alpha = 1/parameters_dot_gradient;
	
    hipblasDscal(handle, n*n, &alpha, auxiliar_matrix, 1);
	
	alpha = 1;
	beta = 1;
	
	hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, &alpha, old_inverse_Hessian, n, 
                &beta, auxiliar_matrix, n, old_inverse_Hessian, n);

	alpha = 0;
	
    hipblasDscal(handle, n*n, &alpha, auxiliar_matrix, 1);
	
	alpha = 1;
	
	hipblasDger(handle, n, n, &alpha, old_gradient, 1, old_gradient, 1, auxiliar_matrix, n);
	
	alpha = 1;
    beta = -1/gradient_dot_Hessian_dot_gradient;
	
	hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, &alpha, old_inverse_Hessian, n, 
                &beta, auxiliar_matrix, n, old_inverse_Hessian, n);

    hipMemcpy(inverse_Hessian_host, old_inverse_Hessian, n*n*sizeof(double), hipMemcpyDeviceToHost);
}

void bfgsInverseHessian(
        double* old_parameters, double* parameters,
        double* old_gradient, double* gradient,
        double* old_inverse_Hessian, double* auxiliar_matrix,
        double* inverse_Hessian_host, int n)
{
	double alpha;
	double beta;
	
	double parameters_dot_gradient;
	double gradient_dot_Hessian_dot_gradient;
	
	hipblasHandle_t handle;
	
	createHandle(&handle);
	
	alpha = -1;
	
    hipblasDaxpy(handle, n, &alpha, old_parameters, 1, parameters, 1);
    hipblasDaxpy(handle, n, &alpha, old_gradient, 1, gradient, 1);
	
    hipblasDdot(handle, n, parameters, 1, gradient, 1, &parameters_dot_gradient);
	
	alpha = 1;
	beta = 0;
	
	hipblasDgemv(handle, HIPBLAS_OP_N, n, n, &alpha, old_inverse_Hessian, n, gradient, 1, &beta, old_gradient, 1);
	
    hipblasDdot(handle, n, gradient, 1, old_gradient, 1, &gradient_dot_Hessian_dot_gradient);
	
	alpha = 0;
	
    hipblasDscal(handle, n*n, &alpha, auxiliar_matrix, 1);
	
	alpha = 1;
	
	hipblasDger(handle, n, n, &alpha, parameters, 1, parameters, 1, auxiliar_matrix, n);
	
	alpha = 1/parameters_dot_gradient;
	
    hipblasDscal(handle, n*n, &alpha, auxiliar_matrix, 1);
	
	alpha = 1;
	beta = 1;
	
    hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, &alpha, old_inverse_Hessian, n,
                &beta, auxiliar_matrix, n, old_inverse_Hessian, n);
				
	alpha = 0;
	
    hipblasDscal(handle, n*n, &alpha, auxiliar_matrix, 1);
	
	alpha = 1;
	
	hipblasDger(handle, n, n, &alpha, old_gradient, 1, old_gradient, 1, auxiliar_matrix, n);
	
	alpha = 1;
    beta = -1/gradient_dot_Hessian_dot_gradient;

    hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, &alpha, old_inverse_Hessian, n,
                &beta, auxiliar_matrix, n, old_inverse_Hessian, n);
				
	alpha = 1/parameters_dot_gradient;
	beta = -1/gradient_dot_Hessian_dot_gradient;
	
    hipblasDscal(handle, n, &alpha, parameters, 1);
	
    hipblasDaxpy(handle, n, &beta, old_gradient, 1, parameters, 1);
	
	alpha = 0;
	
    hipblasDscal(handle, n*n, &alpha, auxiliar_matrix, 1);
	
	hipblasDger(handle, n, n, &gradient_dot_Hessian_dot_gradient, parameters, 1, parameters, 1, auxiliar_matrix, n);
	
	alpha = 1;
	beta = 1;
	
    hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, &alpha, old_inverse_Hessian, n,
                &beta, auxiliar_matrix, n, old_inverse_Hessian, n);
				
    hipMemcpy(inverse_Hessian_host, old_inverse_Hessian, n*n*sizeof(double), hipMemcpyDeviceToHost);
}
